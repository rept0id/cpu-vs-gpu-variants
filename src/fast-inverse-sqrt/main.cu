#include <iostream>
#include <cmath>
#include <chrono>
#include <vector>
#include <hip/hip_runtime.h>

// Fast Inverse Square Root

// Fast Inverse Square Root : CPU
void cpuFisqrt(const std::vector<float>& _arr, std::vector<float>& _result) {
    for (size_t idx = 0; idx < _arr.size(); ++idx) {
        float num = _arr[idx];

        /*** * * ***/

        // FISQRT

        int32_t i;
        float x2, y;
        const float threehalfs = 1.5f;

        x2 = num * 0.5f;
        y = num;
        // FISQRT : Bit-level floating-point hack
        i = *reinterpret_cast<int32_t*>(&y);
        // FISQRT : Magic number for the approximation
        i = 0x5f3759df - (i >> 1);
        y = *reinterpret_cast<float*>(&i);
        
        // 1st iteration of Newton's method (FISQRT)
        y = y * (threehalfs - (x2 * y * y));     
        
        // 2nd iteration of Newton's method for better accuracy (FISQRT)
        y = y * (threehalfs - (x2 * y * y));   

        /*** * * ***/  

        _result[idx] = y; // Store the result
    }
}

// Fast Inverse Square Root : GPU 

// Fast Inverse Square Root : GPU : CUDA

// Fast Inverse Square Root : GPU : CUDA : kernel
__global__ void CudaGpuFisqrt(float* _arr, float* _result, size_t size) {
    int idx;

    /*** * * ***/

    idx = blockIdx.x * blockDim.x + threadIdx.x;

    /*** * * ***/

    if (idx < size) {
        float num = _arr[idx];

        /*** * * ***/

        // FISQRT

        int32_t i;
        float x2, y;
        const float threehalfs = 1.5f;

        x2 = num * 0.5f;
        y = num;
        // FISQRT : Bit-level floating-point hack
        i = *reinterpret_cast<int32_t*>(&y);
        // FISQRT : Magic number for the approximation
        i = 0x5f3759df - (i >> 1);
        y = *reinterpret_cast<float*>(&i);
        
        // 1st iteration of Newton's method (FISQRT)
        y = y * (threehalfs - (x2 * y * y));                
        
        // 2nd iteration of Newton's method for better accuracy (FISQRT)
        y = y * (threehalfs - (x2 * y * y));                

        /*** * * ***/

        _result[idx] = y;
    }
}


// Benchmark

// Benchmark : CPU

// Benchmark : CPU : Fast Inverse Square Root

int cpuFisqrtBenchmark(const std::vector<float>& _req, const int timeoutSec) {
    int count = 0;

    std::vector<float> _res(_req.size());

    std::chrono::_V2::system_clock::time_point startChrono;

    /*** * * ***/

    startChrono = std::chrono::high_resolution_clock::now();
    while (std::chrono::high_resolution_clock::now() - startChrono < std::chrono::seconds(timeoutSec)) {
        cpuFisqrt(_req, _res);
        count++;
    }

    /*** * * ***/

    return count;
}

int cudaGpuFisqrtBenchmark(const std::vector<float>& _req, const int timeoutSec) {
    int count = 0;

    float *_reqGpu;
    float *_resGpu;

    std::chrono::_V2::system_clock::time_point startChrono;

    int blockSize;
    int gridSize;

    /*** * * ***/

    // Allocate memory on GPU
    hipMalloc(&_reqGpu, _req.size() * sizeof(float));
    hipMalloc(&_resGpu, _req.size() * sizeof(float));

    // Copy data to GPU
    hipMemcpy(_reqGpu, _req.data(), _req.size() * sizeof(float), hipMemcpyHostToDevice);

    startChrono = std::chrono::high_resolution_clock::now();
    while (std::chrono::high_resolution_clock::now() - startChrono < std::chrono::seconds(timeoutSec)) {
        blockSize = 256;
        gridSize = (_req.size() + blockSize - 1) / blockSize;

        CudaGpuFisqrt<<<gridSize, blockSize>>>(_reqGpu, _resGpu, _req.size());
        hipDeviceSynchronize();

        count++;
    }

    // Free GPU memory
    hipFree(_reqGpu);
    hipFree(_resGpu);

    return count;
}

/*** * * ***/

int main() {
    std::vector<float> _req(10'000'000);

    /*** * * ***/

    for (size_t i = 0; i < _req.size(); ++i) {
        _req[i] = static_cast<float>(rand()) / RAND_MAX;
    }

    /*** * * ***/

    // Benchmarks

    // Benchmarks : CPU

    std::cout << "CPU calculations in one second: ";
    std::cout << cpuFisqrtBenchmark(_req, 1);
    std::cout << std::endl;

    // Benchmarks : GPU (CUDA)

    std::cout << "CUDA calculations in one second: ";
    std::cout << cudaGpuFisqrtBenchmark(_req, 1);
    std::cout << std::endl;

    /*** * * ***/

    return 0;
}

